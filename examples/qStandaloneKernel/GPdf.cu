#include "hip/hip_runtime.h"
#include "../../GlobalCudaDefines.hh"
#include "GPdf.hh" 
#include "thrust/sequence.h" 
#include "thrust/iterator/constant_iterator.h" 
#include <fstream> 

// LANDAU pdf : algorithm from CERNLIB G110 denlan
// same algorithm is used in GSL

MEM_CONSTANT fptype p1[5] = {0.4259894875,-0.1249762550, 0.03984243700, -0.006298287635,   0.001511162253};
MEM_CONSTANT fptype q1[5] = {1.0         ,-0.3388260629, 0.09594393323, -0.01608042283,    0.003778942063};

MEM_CONSTANT fptype p2[5] = {0.1788541609, 0.1173957403, 0.01488850518, -0.001394989411,   0.0001283617211};
MEM_CONSTANT fptype q2[5] = {1.0         , 0.7428795082, 0.3153932961,   0.06694219548,    0.008790609714};

MEM_CONSTANT fptype p3[5] = {0.1788544503, 0.09359161662,0.006325387654, 0.00006611667319,-0.000002031049101};
MEM_CONSTANT fptype q3[5] = {1.0         , 0.6097809921, 0.2560616665,   0.04746722384,    0.006957301675};

MEM_CONSTANT fptype p4[5] = {0.9874054407, 118.6723273,  849.2794360,   -743.7792444,      427.0262186};
MEM_CONSTANT fptype q4[5] = {1.0         , 106.8615961,  337.6496214,    2016.712389,      1597.063511};

MEM_CONSTANT fptype p5[5] = {1.003675074,  167.5702434,  4789.711289,    21217.86767,     -22324.94910};
MEM_CONSTANT fptype q5[5] = {1.0         , 156.9424537,  3745.310488,    9834.698876,      66924.28357};

MEM_CONSTANT fptype p6[5] = {1.000827619,  664.9143136,  62972.92665,    475554.6998,     -5743609.109};
MEM_CONSTANT fptype q6[5] = {1.0         , 651.4101098,  56974.73333,    165917.4725,     -2815759.939};

MEM_CONSTANT fptype a1[3] = {0.04166666667,-0.01996527778, 0.02709538966};
MEM_CONSTANT fptype a2[2] = {-1.845568670,-4.284640743};

MEM_CONSTANT fptype mpv;
MEM_CONSTANT fptype sigma;

__device__ fptype device_Landau (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[0];

  if (sigma <= 0) return 0;

  fptype v = (x - mpv)/sigma;

  fptype u, ue, us, denlan;
  if (v < -5.5) {
    u   = EXP(v+1.0);
    if (u < 1e-10) return 0.0;
    ue  = EXP(-1/u);
    us  = SQRT(u);
    denlan = 0.3989422803*(ue/us)*(1+(a1[0]+(a1[1]+a1[2]*u)*u)*u);
  }
  else if (v < -1) {
    u   = EXP(-v-1);
    denlan = EXP(-u)*SQRT(u)*
      (p1[0]+(p1[1]+(p1[2]+(p1[3]+p1[4]*v)*v)*v)*v)/
      (q1[0]+(q1[1]+(q1[2]+(q1[3]+q1[4]*v)*v)*v)*v);
  }
  else if (v < 1) {
    denlan = (p2[0]+(p2[1]+(p2[2]+(p2[3]+p2[4]*v)*v)*v)*v)/
      (q2[0]+(q2[1]+(q2[2]+(q2[3]+q2[4]*v)*v)*v)*v);
  }
  else if (v < 5) {
    denlan = (p3[0]+(p3[1]+(p3[2]+(p3[3]+p3[4]*v)*v)*v)*v)/
      (q3[0]+(q3[1]+(q3[2]+(q3[3]+q3[4]*v)*v)*v)*v);
  }
  else if (v < 12) {
    u   = 1/v;
    denlan = u*u*(p4[0]+(p4[1]+(p4[2]+(p4[3]+p4[4]*u)*u)*u)*u)/
      (q4[0]+(q4[1]+(q4[2]+(q4[3]+q4[4]*u)*u)*u)*u);
  }
  else if (v < 50) {
    u   = 1/v;
    denlan = u*u*(p5[0]+(p5[1]+(p5[2]+(p5[3]+p5[4]*u)*u)*u)*u)/
      (q5[0]+(q5[1]+(q5[2]+(q5[3]+q5[4]*u)*u)*u)*u);
  }
  else if (v < 300) {
    u   = 1/v;
    denlan = u*u*(p6[0]+(p6[1]+(p6[2]+(p6[3]+p6[4]*u)*u)*u)*u)/
      (q6[0]+(q6[1]+(q6[2]+(q6[3]+q6[4]*u)*u)*u)*u);
  }
  else {
    u   = 1/(v-v*std::log(v)/(v+1));
    denlan = u*u*(1+(a2[0]+a2[1]*u)*u);
  }
  return denlan/sigma;
}

MEM_DEVICE device_function_ptr ptr_to_Landau = device_Landau;

// These variables are either function-pointer related (thus specific to this implementation)
// or constrained to be in the CUDAglob translation unit by nvcc limitations; otherwise they 
// would be in PdfBase. 

// Device-side, translation-unit constrained. 
MEM_CONSTANT fptype hipArray[maxParams];           // Holds device-side fit parameters. 
MEM_CONSTANT unsigned int paramIndices[maxParams];  // Holds functor-specific indices into hipArray. Also overloaded to hold integer constants (ie parameters that cannot vary.) 
MEM_CONSTANT fptype functorConstants[maxParams];    // Holds non-integer constants. Notice that first entry is number of events. 
MEM_CONSTANT fptype normalisationFactors[maxParams]; 

// For debugging 
MEM_CONSTANT int callnumber; 
MEM_CONSTANT int gpuDebug; 
MEM_CONSTANT unsigned int debugParamIndex;
MEM_DEVICE int internalDebug1 = -1; 
MEM_DEVICE int internalDebug2 = -1; 
MEM_DEVICE int internalDebug3 = -1; 
int cpuDebug = 0; 
#ifdef PROFILING
MEM_DEVICE fptype timeHistogram[10000]; 
fptype host_timeHist[10000];
#endif 

// Function-pointer related. 
MEM_DEVICE void* device_function_table[200]; // Not clear why this cannot be MEM_CONSTANT, but it causes crashes to declare it so. 
void* host_function_table[200];
unsigned int num_device_functions = 0; 
map<void*, int> functionAddressToDeviceIndexMap; 

// For use in debugging memory issues
void printMemoryStatus (std::string file, int line) {
  size_t memfree = 0;
  size_t memtotal = 0; 
  SYNCH(); 
// Thrust 1.7 will make the use of THRUST_DEVICE_BACKEND an error
#if THRUST_DEVICE_BACKEND==THRUST_DEVICE_BACKEND_OMP || THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
#else
  hipMemGetInfo(&memfree, &memtotal); 
#endif
  SYNCH(); 
  std::cout << "Memory status " << file << " " << line << " Free " << memfree << " Total " << memtotal << " Used " << (memtotal - memfree) << std::endl;
}


#include <execinfo.h>
void* stackarray[10];
void abortWithCudaPrintFlush (std::string file, int line, std::string reason, const PdfBase* pdf = 0) {
#ifdef CUDAPRINT
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
#endif
  std::cout << "Abort called from " << file << " line " << line << " due to " << reason << std::endl; 
  if (pdf) {
    PdfBase::parCont pars;
    pdf->getParameters(pars);
    std::cout << "Parameters of " << pdf->getName() << " : \n";
    for (PdfBase::parIter v = pars.begin(); v != pars.end(); ++v) {
      if (0 > (*v)->index) continue; 
      std::cout << "  " << (*v)->name << " (" << (*v)->index << ") :\t" << host_params[(*v)->index] << std::endl;
    }
  }

  std::cout << "Parameters (" << totalParams << ") :\n"; 
  for (int i = 0; i < totalParams; ++i) {
    std::cout << host_params[i] << " ";
  }
  std::cout << std::endl; 


  // get void* pointers for all entries on the stack
  size_t size = backtrace(stackarray, 10);
  // print out all the frames to stderr
  backtrace_symbols_fd(stackarray, size, 2);

  exit(1); 
}

/*
EXEC_TARGET fptype calculateEval (fptype rawPdf, fptype* evtVal, unsigned int par) {
  // Just return the raw PDF value, for use in (eg) normalisation. 
  return rawPdf; 
}

EXEC_TARGET fptype calculateNLL (fptype rawPdf, fptype* evtVal, unsigned int par) {
  //if ((10 > callnumber) && (THREADIDX < 10) && (BLOCKIDX == 0)) cuPrintf("calculateNll %i %f %f %f\n", callnumber, rawPdf, normalisationFactors[par], rawPdf*normalisationFactors[par]);
  //if (THREADIDX < 50) printf("Thread %i %f %f\n", THREADIDX, rawPdf, normalisationFactors[par]); 
  rawPdf *= normalisationFactors[par];
  return rawPdf > 0 ? -LOG(rawPdf) : 0; 
}

EXEC_TARGET fptype calculateProb (fptype rawPdf, fptype* evtVal, unsigned int par) {
  // Return probability, ie normalised PDF value.
  return rawPdf * normalisationFactors[par];
}

EXEC_TARGET fptype calculateBinAvg (fptype rawPdf, fptype* evtVal, unsigned int par) {
  rawPdf *= normalisationFactors[par];
  rawPdf *= evtVal[1]; // Bin volume 
  // Log-likelihood of numEvents with expectation of exp is (-exp + numEvents*ln(exp) - ln(numEvents!)). 
  // The last is constant, so we drop it; and then multiply by minus one to get the negative log-likelihood. 
  if (rawPdf > 0) {
    fptype expEvents = functorConstants[0]*rawPdf;
    return (expEvents - evtVal[0]*log(expEvents)); 
  }
  return 0; 
}

EXEC_TARGET fptype calculateBinWithError (fptype rawPdf, fptype* evtVal, unsigned int par) {
  // In this case interpret the rawPdf as just a number, not a number of events. 
  // Do not divide by integral over phase space, do not multiply by bin volume, 
  // and do not collect 200 dollars. evtVal should have the structure (bin entry, bin error). 
  //printf("[%i, %i] ((%f - %f) / %f)^2 = %f\n", BLOCKIDX, THREADIDX, rawPdf, evtVal[0], evtVal[1], POW((rawPdf - evtVal[0]) / evtVal[1], 2)); 
  rawPdf -= evtVal[0]; // Subtract observed value.
  rawPdf /= evtVal[1]; // Divide by error.
  rawPdf *= rawPdf; 
  return rawPdf; 
}

EXEC_TARGET fptype calculateChisq (fptype rawPdf, fptype* evtVal, unsigned int par) {
  rawPdf *= normalisationFactors[par];
  rawPdf *= evtVal[1]; // Bin volume 

  return pow(rawPdf * functorConstants[0] - evtVal[0], 2) / (evtVal[0] > 1 ? evtVal[0] : 1); 
}

MEM_DEVICE device_metric_ptr ptr_to_Eval         = calculateEval; 
MEM_DEVICE device_metric_ptr ptr_to_NLL          = calculateNLL;  
MEM_DEVICE device_metric_ptr ptr_to_Prob         = calculateProb; 
MEM_DEVICE device_metric_ptr ptr_to_BinAvg       = calculateBinAvg;  
MEM_DEVICE device_metric_ptr ptr_to_BinWithError = calculateBinWithError;
MEM_DEVICE device_metric_ptr ptr_to_Chisq        = calculateChisq; 
*/

void* host_fcn_ptr = 0;

/*
void* getMetricPointer (std::string name) {
  #define CHOOSE_PTR(ptrname) if (name == #ptrname) GET_FUNCTION_ADDR(ptrname);
  host_fcn_ptr = 0; 
  CHOOSE_PTR(ptr_to_Eval); 
  CHOOSE_PTR(ptr_to_NLL); 
  CHOOSE_PTR(ptr_to_Prob); 
  CHOOSE_PTR(ptr_to_BinAvg); 
  CHOOSE_PTR(ptr_to_BinWithError); 
  CHOOSE_PTR(ptr_to_Chisq); 

  assert(host_fcn_ptr); 

  return host_fcn_ptr;
#undef CHOOSE_PTR
}
*/

GPdf::GPdf (Variable* x, std::string n, Variable *m, Variable *s) 
  : PdfBase(x, n)
  , logger(0)
{
  //std::cout << "Created " << n << std::endl; 

  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(m));
  pindices.push_back(registerParameter(s));
  GET_FUNCTION_ADDR(ptr_to_Landau);
  initialise(pindices);

  setMetrics ();
}

__host__ int GPdf::findFunctionIdx (void* dev_functionPtr) {
  // Code specific to function-pointer implementation 
  map<void*, int>::iterator localPos = functionAddressToDeviceIndexMap.find(dev_functionPtr); 
  if (localPos != functionAddressToDeviceIndexMap.end()) {
    return (*localPos).second; 
  }

  int fIdx = num_device_functions;   
  host_function_table[num_device_functions] = dev_functionPtr;
  functionAddressToDeviceIndexMap[dev_functionPtr] = num_device_functions; 
  num_device_functions++; 
  MEMCPY_TO_SYMBOL(device_function_table, host_function_table, num_device_functions*sizeof(void*), 0, hipMemcpyHostToDevice); 

#ifdef PROFILING
  host_timeHist[fIdx] = 0; 
  MEMCPY_TO_SYMBOL(timeHistogram, host_timeHist, 10000*sizeof(fptype), 0);
#endif 

  return fIdx; 
}

__host__ void GPdf::initialise (std::vector<unsigned int> pindices, void* dev_functionPtr) {
  if (!fitControl) setFitControl(new UnbinnedNllFit()); 

  // MetricTaker must be created after PdfBase initialisation is done.
  PdfBase::initialiseIndices(pindices); 

  functionIdx = findFunctionIdx(dev_functionPtr); 
  setMetrics(); 
}

__host__ void GPdf::setDebugMask (int mask, bool setSpecific) const {
  cpuDebug = mask; 
#if THRUST_DEVICE_BACKEND==THRUST_DEVICE_BACKEND_OMP
  gpuDebug = cpuDebug;
  if (setSpecific) debugParamIndex = parameters; 
#else
  MEMCPY_TO_SYMBOL(gpuDebug, &cpuDebug, sizeof(int), 0, hipMemcpyHostToDevice);
  if (setSpecific) MEMCPY_TO_SYMBOL(debugParamIndex, &parameters, sizeof(unsigned int), 0, hipMemcpyHostToDevice);
#endif
} 

__host__ void GPdf::setMetrics () {
  //if (logger) delete logger;
  //logger = new MetricTaker(this, getMetricPointer(fitControl->getMetric()));  

  if (logger) delete logger;
  logger = new MetricTakerKnown();
}

__host__ double GPdf::sumOfNll (int numVars) const {
  //printf ("sumOfNll\n");
  static thrust::plus<double> cudaPlus;
  thrust::constant_iterator<int> eventSize(numVars); 
  thrust::constant_iterator<fptype*> arrayAddress(dev_event_array); 
  double dummy = 0;

  //if (host_callnumber >= 2) abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " debug abort", this); 
  thrust::counting_iterator<int> eventIndex(0); 
  return thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
				  thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
				  *logger, dummy, cudaPlus);   
}

__host__ double GPdf::calculateNLL () const {
  if (cpuDebug & 1) std::cout << getName() << " entering calculateNLL (" << host_callnumber << ")" << std::endl; 

  //manually copy params into values?
  Variable *m = getParameterByName ("mpv");
  Variable *s = getParameterByName ("sigma");

  //printf ("m->mixValue:%f\n", m->mixValue);
  //printf ("s->mixValue:%f\n", s->mixValue);

  hipMemcpyToSymbol(HIP_SYMBOL(mpv), &m->mixValue, sizeof (fptype));
  hipMemcpyToSymbol(HIP_SYMBOL(sigma), &s->mixValue, sizeof(fptype));

  //MEMCPY_TO_SYMBOL(callnumber, &host_callnumber, sizeof(int)); 
  //int oldMask = cpuDebug; 
  //if (0 == host_callnumber) setDebugMask(0, false); 
  //std::cout << "Start norm " << getName() << std::endl;
  normalise();
  //std::cout << "Norm done\n"; 
  //if ((0 == host_callnumber) && (1 == oldMask)) setDebugMask(1, false); 

  
  //if (cpuDebug & 1) {
  //std::cout << "Norm factors: ";
  //for (int i = 0; i < totalParams; ++i) std::cout << host_normalisation[i] << " ";
  //std::cout << std::endl;
  //} 
   
  if (host_normalisation[parameters] <= 0) 
    abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " non-positive normalisation", this);

  MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  SYNCH(); // Ensure normalisation integrals are finished

  int numVars = observables.size(); 
  if (fitControl->binnedFit()) {
    numVars += 2;
    numVars *= -1; 
  }

  fptype ret = sumOfNll(numVars); 
  if (0 == ret) abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " zero NLL", this); 
  //if (cpuDebug & 1) std::cout << "Full NLL " << host_callnumber << " : " << 2*ret << std::endl;

  //setDebugMask(0); 

  //if ((cpuDebug & 1) && (host_callnumber >= 1)) abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " debug abort", this); 
  return 2*ret; 
}

__host__ void GPdf::evaluateAtPoints (Variable* var, std::vector<fptype>& res) {
  // NB: This does not project correctly in multidimensional datasets, because all observables
  // other than 'var' will have, for every event, whatever value they happened to get set to last
  // time they were set. This is likely to be the value from the last event in whatever dataset  
  // you were fitting to, but at any rate you don't get the probability-weighted integral over
  // the other observables. 

  copyParams(); 
  normalise(); 

  MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  UnbinnedDataSet tempdata(observables);

  double step = (var->upperlimit - var->lowerlimit) / var->numbins; 
  for (int i = 0; i < var->numbins; ++i) {
    var->value = var->lowerlimit + (i+0.5)*step;
    tempdata.addEvent(); 
  }
  setData(&tempdata);

  thrust::counting_iterator<int> eventIndex(0); 
  thrust::constant_iterator<int> eventSize(observables.size()); 
  thrust::constant_iterator<fptype*> arrayAddress(dev_event_array); 
  thrust::device_vector<fptype> results(var->numbins); 

/*
  //MetricTaker evalor(this, getMetricPointer("ptr_to_Eval")); 
#ifdef TARGET_MPI
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + m_iEventsPerTask, arrayAddress, eventSize)),
		    results.begin(),
		    evalor); 
#else
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
		    results.begin(),
		    evalor); 
#endif
  thrust::host_vector<fptype> h_results = results;
  res.clear();
  res.resize(var->numbins);
  for (int i = 0; i < var->numbins; ++i) {
    res[i] = h_results[i] * host_normalisation[parameters];
  }
*/
}

__host__ void GPdf::evaluateAtPoints (std::vector<fptype>& points) const {
  /*
  std::set<Variable*> vars;
  getParameters(vars);
  unsigned int maxIndex = 0;
  for (std::set<Variable*>::iterator i = vars.begin(); i != vars.end(); ++i) {
    if ((*i)->getIndex() < maxIndex) continue;
    maxIndex = (*i)->getIndex();
  }
  std::vector<double> params;
  params.resize(maxIndex+1);
  for (std::set<Variable*>::iterator i = vars.begin(); i != vars.end(); ++i) {
    if (0 > (*i)->getIndex()) continue;
    params[(*i)->getIndex()] = (*i)->value;
  } 
  copyParams(params); 

  thrust::device_vector<fptype> d_vec = points; 
  normalise(); 
  MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  thrust::transform(d_vec.begin(), d_vec.end(), d_vec.begin(), *evalor);
  thrust::host_vector<fptype> h_vec = d_vec;
  for (unsigned int i = 0; i < points.size(); ++i) points[i] = h_vec[i]; 
  */
}

__host__ void GPdf::scan (Variable* var, std::vector<fptype>& values) {
  fptype step = var->upperlimit;
  step -= var->lowerlimit;
  step /= var->numbins;
  values.clear(); 
  for (fptype v = var->lowerlimit + 0.5*step; v < var->upperlimit; v += step) {
    var->value = v;
    copyParams();
    fptype curr = calculateNLL(); 
    values.push_back(curr);
  }
}

__host__ void GPdf::setParameterConstantness (bool constant) {
  PdfBase::parCont pars; 
  getParameters(pars); 
  for (PdfBase::parIter p = pars.begin(); p != pars.end(); ++p) {
    (*p)->fixed = constant; 
  }
}

__host__ fptype GPdf::getValue () {
  // Returns the value of the PDF at a single point. 
  // Execute redundantly in all threads for OpenMP multiGPU case
  copyParams(); 
  normalise(); 
  MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 

  UnbinnedDataSet point(observables); 
  point.addEvent(); 
  setData(&point); 

  thrust::counting_iterator<int> eventIndex(0); 
  thrust::constant_iterator<int> eventSize(observables.size()); 
  thrust::constant_iterator<fptype*> arrayAddress(dev_event_array); 
  thrust::device_vector<fptype> results(1); 

/*
  MetricTaker evalor(this, getMetricPointer("ptr_to_Eval"));
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + 1, arrayAddress, eventSize)),
		    results.begin(),
		    evalor); 
*/
  return results[0];
}

__host__ fptype GPdf::normalise () const {
  //if (cpuDebug & 1) std::cout << "Normalising " << getName() << " " << hasAnalyticIntegral() << " " << normRanges << std::endl;

  if (!fitControl->metricIsPdf()) {
    host_normalisation[parameters] = 1.0; 
    return 1.0;
  }

  fptype ret = 1;
  if (hasAnalyticIntegral()) {
    for (obsConstIter v = obsCBegin(); v != obsCEnd(); ++v) { // Loop goes only over observables of this PDF. 
      //if (cpuDebug & 1) std::cout << "Analytically integrating " << getName() << " over " << (*v)->name << std::endl; 
      ret *= integrate((*v)->lowerlimit, (*v)->upperlimit);
    }
    host_normalisation[parameters] = 1.0/ret;
    //if (cpuDebug & 1) std::cout << "Analytic integral of " << getName() << " is " << ret << std::endl; 
    return ret; 
  } 

  int totalBins = 1; 
  for (obsConstIter v = obsCBegin(); v != obsCEnd(); ++v) {
    ret *= ((*v)->upperlimit - (*v)->lowerlimit);
    totalBins *= (integrationBins > 0 ? integrationBins : (*v)->numbins); 
    //if (cpuDebug & 1) std::cout << "Total bins " << totalBins << " due to " << (*v)->name << " " << integrationBins << " " << (*v)->numbins << std::endl; 
  }
  ret /= totalBins; 

  fptype dummy = 0; 
  static thrust::plus<fptype> cudaPlus;
  thrust::constant_iterator<fptype*> arrayAddress(normRanges); 
  thrust::constant_iterator<int> eventSize(observables.size());
  thrust::counting_iterator<int> binIndex(0); 
  fptype sum = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(binIndex, eventSize, arrayAddress)),
					thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, eventSize, arrayAddress)),
					*logger, dummy, cudaPlus); 

  //std::cout << "sum: " << sum << " mpv: " << getParameterByName("mpv")->value << " sigma: " << getParameterByName ("sigma")->value << std::endl;

  if (std::isnan(sum)) {
    abortWithCudaPrintFlush(__FILE__, __LINE__, getName() + " NaN in normalisation", this); 
  }
  else if (0 >= sum) { 
    abortWithCudaPrintFlush(__FILE__, __LINE__, "Non-positive normalisation", this); 
  }
  ret *= sum;
  if (0 == ret) abortWithCudaPrintFlush(__FILE__, __LINE__, "Zero integral"); 
  host_normalisation[parameters] = 1.0/ret;

  return (fptype) ret; 
}

#ifdef PROFILING
MEM_CONSTANT fptype conversion = (1.0 / CLOCKS_PER_SEC); 
EXEC_TARGET fptype callFunction (fptype* eventAddress, unsigned int functionIdx, unsigned int paramIdx) {
  clock_t start = clock();
  fptype ret = (*(reinterpret_cast<device_function_ptr>(device_function_table[functionIdx])))(eventAddress, hipArray, paramIndices + paramIdx);
  clock_t stop = clock(); 
  if ((0 == THREADIDX + BLOCKIDX) && (stop > start)) {
    // Avoid issue when stop overflows and start doesn't. 
    timeHistogram[functionIdx*100 + paramIdx] += ((stop - start) * conversion); 
    //printf("Clock: %li %li %li | %u %f\n", (long) start, (long) stop, (long) (stop - start), functionIdx, timeHistogram[functionIdx]); 
  }

  return ret; 
}
#else 
EXEC_TARGET fptype callFunction (fptype* eventAddress, unsigned int functionIdx, unsigned int paramIdx) {
  return (*(reinterpret_cast<device_function_ptr>(device_function_table[functionIdx])))(eventAddress, hipArray, paramIndices + paramIdx);
}
#endif 

// Notice that operators are distinguished by the order of the operands,
// and not otherwise! It's up to the user to make his tuples correctly. 

// Main operator: Calls the PDF to get a predicted value, then the metric 
// to get the goodness-of-prediction number which is returned to MINUIT. 
/*
EXEC_TARGET fptype MetricTaker::operator () (thrust::tuple<int, fptype*, int> t) const
{
  // Calculate event offset for this thread. 
  int eventIndex = thrust::get<0>(t);
  int eventSize  = thrust::get<2>(t);
  fptype* eventAddress = thrust::get<1>(t) + (eventIndex * abs(eventSize)); 

  // Causes stack size to be statically undeterminable.
  fptype ret = callFunction(eventAddress, functionIdx, parameters);

  // Notice assumption here! For unbinned fits the 'eventAddress' pointer won't be used
  // in the metric, so it doesn't matter what it is. For binned fits it is assumed that
  // the structure of the event is (obs1 obs2... binentry binvolume), so that the array
  // passed to the metric consists of (binentry binvolume). 
  ret = (*(reinterpret_cast<device_metric_ptr>(device_function_table[metricIndex])))(ret, eventAddress + (abs(eventSize)-2), parameters);
  return ret; 
}
*/

EXEC_TARGET fptype MetricTakerKnown::operator () (thrust::tuple<int, fptype*, int> t) const
{
  int eventIndex = thrust::get<0>(t);
  int eventSize = thrust::get<2>(t);

  fptype *eventAddress = thrust::get<1>(t) + (eventIndex * abs(eventSize));

  fptype *functionIdx = NULL;
  unsigned int *pIdx = NULL;
  fptype ret = device_Landau(eventAddress, functionIdx, pIdx);

  fptype r2 = ret *= normalisationFactors[0];
  
  return r2 > 0 ? -LOG(r2) : 0.0;
}
 
// Operator for binned evaluation, no metric. 
// Used in normalisation. 
/*
#define MAX_NUM_OBSERVABLES 5
EXEC_TARGET fptype MetricTaker::operator () (thrust::tuple<int, int, fptype*> t) const {
  // Bin index, event size, base address [lower, upper, numbins] 
 
  int evtSize = thrust::get<1>(t);
  int binNumber = thrust::get<0>(t);
  
  // Do not understand why this cannot be declared __shared__. Dynamically allocating shared memory is apparently complicated. 
  //fptype* binCenters = (fptype*) malloc(evtSize * sizeof(fptype));
  MEM_SHARED fptype binCenters[1024*MAX_NUM_OBSERVABLES];

  // To convert global bin number to (x,y,z...) coordinates: For each dimension, take the mod 
  // with the number of bins in that dimension. Then divide by the number of bins, in effect
  // collapsing so the grid has one fewer dimension. Rinse and repeat. 
  unsigned int* indices = paramIndices + parameters;
  for (int i = 0; i < evtSize; ++i) {
    fptype lowerBound = thrust::get<2>(t)[3*i+0];
    fptype upperBound = thrust::get<2>(t)[3*i+1];
    int numBins    = (int) FLOOR(thrust::get<2>(t)[3*i+2] + 0.5); 
    int localBin = binNumber % numBins;

    fptype x = upperBound - lowerBound; 
    x /= numBins;
    x *= (localBin + 0.5); 
    x += lowerBound;
    binCenters[indices[indices[0] + 2 + i]+THREADIDX*MAX_NUM_OBSERVABLES] = x; 
    binNumber /= numBins;
  }

  // Causes stack size to be statically undeterminable.
  fptype ret = callFunction(binCenters+THREADIDX*MAX_NUM_OBSERVABLES, functionIdx, parameters); 
  return ret; 
}
*/

EXEC_TARGET fptype MetricTakerKnown::operator () (thrust::tuple<int, int, fptype*> t) const
{
  int evtSize = thrust::get<1>(t);
  int binNumber = thrust::get<0> (t);

  MEM_SHARED fptype binCenters[1024*5];

  for (int i = 0; i < evtSize; ++i)
  {
    fptype lowerBound = thrust::get<2>(t)[3*i+0];
    fptype upperBound = thrust::get<2>(t)[3*i+1];
    int numBins    = (int) FLOOR(thrust::get<2>(t)[3*i+2] + 0.5);
    int localBin = binNumber % numBins;

    fptype x = upperBound - lowerBound;
    x /= numBins;
    x *= (localBin + 0.5);
    x += lowerBound;
    binCenters[THREADIDX*5] = x;
    binNumber /= numBins;
  }

  return device_Landau(binCenters + THREADIDX*5, 0, 0);
}

  //Untested so far, no examples call this function!
__host__ void GPdf::getCompProbsAtDataPoints (std::vector<std::vector<fptype> >& values)
{
  copyParams(); 
  double overall = normalise();
  std::cout << "normalize () - " << overall << std::endl;
  MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 

  int numVars = observables.size(); 
  if (fitControl->binnedFit()) {
    numVars += 2;
    numVars *= -1; 
  }

  thrust::device_vector<fptype> results(numEntries); 
  thrust::constant_iterator<int> eventSize(numVars); 
  thrust::constant_iterator<fptype*> arrayAddress(dev_event_array); 
  thrust::counting_iterator<int> eventIndex(0); 

/*
  MetricTaker evalor(this, getMetricPointer("ptr_to_Prob")); 

#ifdef TARGET_MPI
  //write to results, send to all?
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + m_iEventsPerTask, arrayAddress, eventSize)),
		    results.begin(), 
		    evalor);

  values.clear(); 
  values.resize(components.size() + 1);

  thrust::host_vector<fptype> host_results = results;
  for (unsigned int i = 0; i < host_results.size(); ++i) {
    values[0].push_back(host_results[i]);
  }
#else
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
		    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
		    results.begin(), 
		    evalor); 
  values.clear(); 
  values.resize(components.size() + 1);

  thrust::host_vector<fptype> host_results = results;
  for (unsigned int i = 0; i < host_results.size(); ++i) {
    values[0].push_back(host_results[i]);
  }
#endif 
*/

/*
  for (unsigned int i = 0; i < components.size(); ++i) {
    MetricTaker compevalor(components[i], getMetricPointer("ptr_to_Prob")); 
    thrust::counting_iterator<int> ceventIndex(0);
#ifdef TARGET_MPI 
    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(ceventIndex, arrayAddress, eventSize)),
		      thrust::make_zip_iterator(thrust::make_tuple(ceventIndex + m_iEventsPerTask, arrayAddress, eventSize)),
		      results.begin(), 
		      compevalor); 

    host_results = results;
    for (unsigned int j = 0; j < host_results.size(); ++j) {
      values[1 + i].push_back(host_results[j]); 
    }
#else
    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(ceventIndex, arrayAddress, eventSize)),
		      thrust::make_zip_iterator(thrust::make_tuple(ceventIndex + numEntries, arrayAddress, eventSize)),
		      results.begin(), 
		      compevalor); 

    host_results = results;
    for (unsigned int j = 0; j < host_results.size(); ++j) {
      values[1 + i].push_back(host_results[j]); 
    }
#endif
  }
*/
}

// still need to add OpenMP/multi-GPU code here
__host__ void GPdf::transformGrid (fptype* host_output) { 
  generateNormRange(); 
  //normalise(); 
  int totalBins = 1; 
  for (obsConstIter v = obsCBegin(); v != obsCEnd(); ++v) {
    totalBins *= (*v)->numbins; 
  }

  thrust::constant_iterator<fptype*> arrayAddress(normRanges); 
  thrust::constant_iterator<int> eventSize(observables.size());
  thrust::counting_iterator<int> binIndex(0); 
  thrust::device_vector<fptype> d_vec;
  d_vec.resize(totalBins); 

  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(binIndex, eventSize, arrayAddress)),
		    thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, eventSize, arrayAddress)),
		    d_vec.begin(), 
		    *logger); 

  //Extra copy here, fixme later
  thrust::host_vector<fptype> h_vec = d_vec;
  for (unsigned int i = 0; i < totalBins; ++i) host_output[i] = h_vec[i]; 
}

MetricTakerKnown::MetricTakerKnown()
{
}

/*
MetricTaker::MetricTaker (PdfBase* dat, void* dev_functionPtr) 
  : metricIndex(0)
  , functionIdx(dat->getFunctionIndex())
  , parameters(dat->getParameterIndex())
{
  //std::cout << "MetricTaker constructor with " << functionIdx << std::endl; 

  map<void*, int>::iterator localPos = functionAddressToDeviceIndexMap.find(dev_functionPtr); 
  if (localPos != functionAddressToDeviceIndexMap.end()) {
    metricIndex = (*localPos).second; 
  }
  else {
    metricIndex = num_device_functions; 
    host_function_table[num_device_functions] = dev_functionPtr;
    functionAddressToDeviceIndexMap[dev_functionPtr] = num_device_functions; 
    num_device_functions++; 
    MEMCPY_TO_SYMBOL(device_function_table, host_function_table, num_device_functions*sizeof(void*), 0, hipMemcpyHostToDevice); 
  }
}

MetricTaker::MetricTaker (int fIdx, int pIdx) 
  : metricIndex(0)
  , functionIdx(fIdx)
  , parameters(pIdx)
{
  // This constructor should only be used for binned evaluation, ie for integrals. 
}
*/

__host__ void GPdf::setFitControl (FitControl* const fc, bool takeOwnerShip) {
  for (unsigned int i = 0; i < components.size(); ++i) {
    components[i]->setFitControl(fc, false); 
  }

  if ((fitControl) && (fitControl->getOwner() == this)) {
    delete fitControl; 
  }
  fitControl = fc; 
  if (takeOwnerShip) {
    fitControl->setOwner(this); 
  }
  setMetrics();
}

#include "../../PdfBase.cu" 
